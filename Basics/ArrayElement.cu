
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void fillArray(int* arr) {
	int tid = threadIdx.x;
	arr[tid] = tid * 10;
}

int main() {
	const int num = 5;
	int h_arr[num]; 
	int* d_arr;

	hipMalloc((void**)&d_arr, num * sizeof(int));

	fillArray <<<1, num>>> (d_arr);
	hipMemcpy(h_arr, d_arr, num * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < num; i++) {
		printf("h_arr[%d] = %d\n", i, h_arr[i]);
	}

	hipFree(d_arr);

	return 0;
}
