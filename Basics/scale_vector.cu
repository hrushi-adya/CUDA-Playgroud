
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void scaleArray(int* arr, int num) {
	int tid = threadIdx.x;
	arr[tid] = arr[tid] * num;
}
int main() {
	const int num = 5;
	int h_arr[num] = {1, 2, 3, 4, 5};
	int* d_arr;

	hipMalloc((void**)&d_arr, num * sizeof(int));
	hipMemcpy(d_arr, h_arr, num * sizeof(int), hipMemcpyHostToDevice);

	scaleArray <<<1, num >>>(d_arr, 2);
	
	hipMemcpy(h_arr, d_arr, num * sizeof(int), hipMemcpyDeviceToHost);

	printf("Scaled Array: \n");
	for (int i = 0; i < num; i++) {
		printf("%d \t", h_arr[i]);
	}

	hipFree(d_arr);

	return 0; 
} 

/*
 * Output:
 * Scaled Array:
 * 2       4       6       8       10
 */ 
