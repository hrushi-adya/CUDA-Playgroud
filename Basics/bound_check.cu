#include "hip/hip_runtime.h"
#include<stdio.h>
#inlude<hip/hip_runtime.h>

__global__ void doubleArray(int* arr, int num) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num) {
		arr[tid] = arr[tid] * 2;
	}
}

int main() {
	const int num = 20;
	int arr[num];
	int* d_arr;

	for (int i = 0; i < num; i++) {
		arr[i] = i;
	}

	hipMalloc((void**)&d_arr, num * sizeof(int));
	hipMemcpy(d_arr, arr, num * sizeof(int), hipMemcpyHostToDevice);

	int threadsPerBlock = 8;
	int blocksPerGrid = (num + threadsPerBlock - 1) / threadsPerBlock;

	// Launch kernel to fill the array
	doubleArray<<<blocksPerGrid, threadsPerBlock >>>(d_arr, num);
	hipMemcpy(arr, d_arr, num * sizeof(int), hipMemcpyDeviceToHost);

	printf("Doubled Array:\n");
	for (int i = 0; i < num; i++) {
		printf("%d\t", arr[i]);
	}

	hipFree(d_arr);

	return 0;
}
